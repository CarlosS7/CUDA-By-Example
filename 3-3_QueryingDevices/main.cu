#include "../common/book.h"
#include "hip/hip_runtime.h"

int main(void)
{
	hipDeviceProp_t prop;

	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));	// Get number of CUDA devices

	// Loop through devices
	for (int i = 0; i < count; i++)
	{
		// Get device properties
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));

		// Do something with the available data
		// Print general information
		printf("--- General Information for Device %d ---\n", i);
		printf("Name: %d\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout: ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		// Print memory information
		printf("--- Memory Information for Device %d ---\n", i);
		printf("Total global mem: %ld\n",	prop.totalGlobalMem);
		printf("Total const mem: %ld\n",	prop.totalConstMem);
		printf("Max mem pitch: %ld\n",		prop.memPitch);
		printf("Texture alignment: %ld\n",	prop.textureAlignment);
		printf("\n");

		printf("--- MP Information for Device %d ---\n", i);
		printf("Multiprocessor count: %d\n",	prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n",		prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n",		prop.regsPerBlock);
		printf("Threads in warp: %d\n",			prop.warpSize);
		printf("Max threads per block: %d\n",	prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0],
														prop.maxThreadsDim[1],
														prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n",	prop.maxGridSize[0],
														prop.maxGridSize[1],
														prop.maxGridSize[2]);
		printf("\n");
	}
}