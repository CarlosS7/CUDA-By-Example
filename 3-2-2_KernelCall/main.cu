
#include <hip/hip_runtime.h>
#include <iostream>

// __global__ alerts the compiler that a function should be built to run on a device, not the host
__global__ void kernel(void)
{
}

// This function is built for the host as normal
int main(void)
{
	kernel<<<1,1>>>();	// The angled brackets denote arguments to be passed to the device runtime
	printf("Hello, world!\n");
	return 0;
}