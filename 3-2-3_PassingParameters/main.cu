#include "hip/hip_runtime.h"
#include <iostream>
#include "../common/book.h"

__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}

int main(void)
{
	int c;
	int *dev_c;
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

	add<<<1,1>>>(2, 7, dev_c);
}