#include "hip/hip_runtime.h"
#include <iostream>
#include "../common/book.h"
#include "hip/hip_runtime.h"

// Code to run on the GPU
__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}

int main(void)
{
	int c;
	int *dev_c;

	// Allocate memory on the device
	// HANDLE_ERROR is from book's sample code
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

	add<<<1,1>>>(2, 7, dev_c);

	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
	printf("2 + 7 = %d\n", c);
	hipFree(dev_c);

	return 0;
}