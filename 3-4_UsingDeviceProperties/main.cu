#include "../common/book.h"
#include "hip/hip_runtime.h"

int main(void)
{
	hipDeviceProp_t prop;	// Device properties
	int dev;				// Device ID

	HANDLE_ERROR(hipGetDevice(&dev));
	printf("ID of current CUDA device: %d\n", dev);

	// Fill a CUDA device property structure with the values we need
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 3;
	// Choose a device based on these properties
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
	HANDLE_ERROR(hipSetDevice(dev));
}